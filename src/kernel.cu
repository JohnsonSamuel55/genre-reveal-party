#include "hip/hip_runtime.h"
// -*- mode: c -*-

#include <cstdlib>
#include <stdio.h>
#include <float.h>

#include "point.h"

//Testing for CUDA errors
#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

//Clustering Calculation
__global__
void distances_k(Point* points_d, int points_size, Point* centroids_d, int k) {
    //Get the index for the current point to work with
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    /*
      Potential improvement would be to not interate cluster_id -> k and instead have a separate thread
      per cluster_id. Not sure if this would yield any performance gains though. It also creates race
      conditions that have to be dealt with.
    */

    if(index < points_size) {
        //printf("idx %d\n", index);
        Point* p = &points_d[index];
        for (int cluster_id = 0; cluster_id < k; cluster_id++) {
            Point c = centroids_d[cluster_id];
            double dist = ((c.x - p->x) * (c.x - p->x) + (c.y - p->y) * (c.y - p->y) + (c.z - p->z) * (c.z - p->z));// point_distance(c, *p);
            if (dist < p->min_dist) {
                p->min_dist = dist;
                p->cluster = cluster_id;
            }
        }
    }
}

// Do sum and reduce of the points x, y, and z values. This is done in 1 block, but could
// probably be sped up if a multi block reduction algorithm were implemented
__global__
void sum_reduce_kernel(Point* points_d, int points_size, int* n_points_d, double* sum_x_d, double* sum_y_d, double* sum_z_d, int k) {
    /// initialize variables
    int idx = threadIdx.x;

    /* FIXME
       __shared__ int n_points[blockDim.x][k];
       __shared__ double sum_x[blockDim.x][k];
       __shared__ double sum_y[blockDim.x][k];
       __shared__ double sum_z[blockDim.x][k];
    */

    __shared__ int n_points[256][5];
    __shared__ double sum_x[256][5];
    __shared__ double sum_y[256][5];
    __shared__ double sum_z[256][5];

    for (int cluster_id = 0; cluster_id < k; cluster_id++) {
        n_points[idx][cluster_id] = 0;
        sum_x[idx][cluster_id] = 0;
        sum_y[idx][cluster_id] = 0;
        sum_z[idx][cluster_id] = 0;
    }

    // XXX it hates something in this for loop
    //printf("dim %d  ",blockDim.x);
    // compute the local sum
    for (int i = idx; i < points_size; i += blockDim.x) {
        Point* p = &points_d[i];
        n_points[idx][p->cluster] += 1;
        sum_x[idx][p->cluster] += p->x;
        sum_y[idx][p->cluster] += p->y;
        sum_z[idx][p->cluster] += p->z;

        p->min_dist = DBL_MAX;
    }

    __syncthreads();

    // do the reduction
    for (int size = blockDim.x / 2; size > 0; size /= 2) {
        if (idx < size) {
            for (int cluster_id = 0; cluster_id < k; cluster_id++) {
                n_points[idx][cluster_id] += n_points[idx + size][cluster_id];
                sum_x[idx][cluster_id] += sum_x[idx + size][cluster_id];
                sum_y[idx][cluster_id] += sum_y[idx + size][cluster_id];
                sum_z[idx][cluster_id] += sum_z[idx + size][cluster_id];
            }
        }
        __syncthreads();
    }

    // put into output buffer
    if (idx == 0) {
        for (int cluster_id = 0; cluster_id < k; ++cluster_id) {
            //printf("updating n  points to %d\n",n_points_d[cluster_id]);
            n_points_d[cluster_id] = n_points[0][cluster_id];
            sum_x_d[cluster_id] = sum_x[0][cluster_id];
            sum_y_d[cluster_id] = sum_y[0][cluster_id];
            sum_z_d[cluster_id] = sum_z[0][cluster_id];
        }
    }
}

extern "C" void cuda_setup(Point* points_h, Point** points_d, int points_size, Point** centroids_d, int** n_points_d, double** sum_x_d, double** sum_y_d, double** sum_z_d, int k) {
    //Allocate device pointers and copy them to the device
    checkCuda(hipMalloc((void **) points_d, sizeof(Point)*points_size));
    checkCuda(hipMalloc((void **) centroids_d, sizeof(Point)*k)); // just allocate the memory now, we will memcpy every kernel call
    checkCuda(hipMalloc((void **) n_points_d, sizeof(int)*k));
    checkCuda(hipMalloc((void **) sum_x_d, sizeof(double)*k));
    checkCuda(hipMalloc((void **) sum_y_d, sizeof(double)*k));
    checkCuda(hipMalloc((void **) sum_z_d, sizeof(double)*k));
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipMemcpy(*points_d, points_h, sizeof(Point)*points_size, hipMemcpyHostToDevice));
    checkCuda(hipDeviceSynchronize());
}

extern "C" void cuda_cleanup(Point* points_h, Point* points_d, int points_size, Point* centroids_d, int* n_points_d, double* sum_x_d, double* sum_y_d, double* sum_z_d) {
    //copy device points to host points
    checkCuda(hipMemcpy(points_h, points_d, sizeof(Point)*points_size, hipMemcpyDeviceToHost));
    checkCuda(hipDeviceSynchronize());

    //Free device pointers
    checkCuda(hipFree(points_d));
    checkCuda(hipFree(centroids_d));
    checkCuda(hipFree(n_points_d));
    checkCuda(hipFree(sum_x_d));
    checkCuda(hipFree(sum_y_d));
    checkCuda(hipFree(sum_z_d));
    checkCuda(hipDeviceSynchronize());
}

// Function that launches the CUDA kernel
extern "C" void cuda_distances_kernel(Point* points_d, int points_size, Point* centroids_h, Point* centroids_d, int k) {
    dim3 DimGrid(ceil(points_size/32.0));
    dim3 DimBlock(32);

    // copy the centroids over
    checkCuda(hipMemcpy(centroids_d, centroids_h, sizeof(Point)*k, hipMemcpyHostToDevice));
    checkCuda(hipDeviceSynchronize());

    //Launch kernel
    distances_k<<<DimGrid, DimBlock>>>(points_d, points_size, centroids_d, k);
    checkCuda(hipDeviceSynchronize());
}

extern "C" void cuda_sum_kernel(Point* points_d,
                                int points_size,
                                int* n_points_h,
                                double* sum_x_h,
                                double* sum_y_h,
                                double* sum_z_h,
                                int* n_points_d,
                                double* sum_x_d,
                                double* sum_y_d,
                                double* sum_z_d,
                                int k) {
    int block_size = 256; // only using 1 block so make this bigger

    // do the computation
    checkCuda(hipDeviceSynchronize());
    // we know we get here
    //printf("b4");
//(points_d, points_size, n_points_d, sum_x_d, sum_y_d, sum_z_d, k)
    sum_reduce_kernel<<<1, block_size>>>(points_d, points_size, n_points_d, sum_x_d, sum_y_d, sum_z_d, k);
    //printf("after"); // and here
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipMemcpy(n_points_h, n_points_d, sizeof(int)*k, hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(sum_x_h, sum_x_d, sizeof(double)*k, hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(sum_y_h, sum_y_d, sizeof(double)*k, hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(sum_z_h, sum_z_d, sizeof(double)*k, hipMemcpyDeviceToHost));
    checkCuda(hipDeviceSynchronize());
}
