#include "hip/hip_runtime.h"
// -*- mode: c -*-

#include <cstdlib>
#include <stdio.h>
#include <float.h>

#include "point.h"

//Testing for CUDA errors
#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//Clustering Calculation
__global__ 
void distances_k(Point* points_d, int points_size, Point* centroids_d, int k) {
    //Get the index for the current point to work with
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    /*
      Potential improvement would be to not interate cluster_id -> k and instead have a separate thread
      per cluster_id. Not sure if this would yield any performance gains though. It also creates race
      conditions that have to be dealt with.
     */

    if(index < points_size) {
        //printf("idx %d\n", index);
        Point* p = &points_d[index];
        for (int cluster_id = 0; cluster_id < k; cluster_id++) {
            Point c = centroids_d[cluster_id];
            double dist = ((c.x - p->x) * (c.x - p->x) + (c.y - p->y) * (c.y - p->y) + (c.z - p->z) * (c.z - p->z));// point_distance(c, *p);
            if (dist < p->min_dist) {
                p->min_dist = dist;
                p->cluster = cluster_id;
            }
        }
    }
}

extern "C" void cuda_setup(Point* points_h, Point** points_d, int points_size, Point** centroids_d, int k) {
    //Allocate device pointers and copy them to the device
    checkCuda(hipMalloc((void **) points_d, sizeof(Point)*points_size));
    checkCuda(hipDeviceSynchronize());
    printf("allocating centroids_d\n");
    checkCuda(hipMalloc((void **) centroids_d, sizeof(Point)*k)); // just allocate the memory now, we will memcpy every kernel call
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipMemcpy(*points_d, points_h, sizeof(Point)*points_size, hipMemcpyHostToDevice));
    checkCuda(hipDeviceSynchronize());
}

extern "C" void cuda_cleanup(Point* points_h, Point* points_d, int points_size, Point* centroids_d) {
    //copy device points to host points
    checkCuda(hipMemcpy(points_h, points_d, sizeof(Point)*points_size, hipMemcpyDeviceToHost));
    checkCuda(hipDeviceSynchronize());
    
    //Free device pointers
    checkCuda(hipFree(points_d));
    checkCuda(hipFree(centroids_d));
    checkCuda(hipDeviceSynchronize());
}

// Function that launches the CUDA kernel
extern "C" void cuda_distances_kernel(Point* points_d, int points_size, Point* centroids_h, Point* centroids_d, int k) {
    // XXX: why is it 2d?
    dim3 DimGrid(ceil(points_size/32.0));
    dim3 DimBlock(32);

    // copy the centroids over
    //checkCuda(hipMalloc((void **) &centroids_d, sizeof(Point)*k)); // just allocate the memory now, we will memcpy every kernel call
    checkCuda(hipMemcpy(centroids_d, centroids_h, sizeof(Point)*k, hipMemcpyHostToDevice));
    //checkCuda(hipMemcpy(points_d, points_h, sizeof(Point)*points_size, hipMemcpyHostToDevice));
    checkCuda(hipDeviceSynchronize());

    //Launch kernel
    distances_k<<<DimGrid, DimBlock>>>(points_d, points_size, centroids_d, k);
    //checkCuda(hipFree(centroids_d));
    checkCuda(hipDeviceSynchronize());
}

/*
// Function that launches the CUDA kernel
extern "C" void cuda_Kernel_2(Point* points_h, int points_size, int* n_points, double* sum_x, double* sum_y, double* sum_z, int k) {
    Point* points_d;
    int* n_points_d;
    double* sum_x_d;
    double* sum_y_d;
    double* sum_z_d;
    int new_points_size = sqrt(points_size); 

    //Allocate device pointers and copy them to the device
    checkCuda(hipMalloc((void **) &points_d, sizeof(Point)*points_size));
    checkCuda(hipMalloc((void **) &n_points_d, sizeof(int)*k));
    checkCuda(hipMalloc((void **) &sum_x_d, sizeof(double)*k));
    checkCuda(hipMalloc((void **) &sum_y_d, sizeof(double)*k));
    checkCuda(hipMalloc((void **) &sum_z_d, sizeof(double)*k));
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipMemcpy(points_d, points_h, sizeof(Point)*points_size, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(n_points_d, n_points, sizeof(int)*k, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(sum_x_d, sum_x, sizeof(double)*k, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(sum_y_d, sum_y, sizeof(double)*k, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(sum_z_d, sum_z, sizeof(double)*k, hipMemcpyHostToDevice));
    checkCuda(hipDeviceSynchronize());
    
    dim3 DimGrid(ceil(new_points_size/32.0), ceil(new_points_size/32.0), 1);
    dim3 DimBlock(32, 32, 1);

    //Launch the kernel
    my_Kernel_2<<<DimGrid, DimBlock>>>(DBL_MAX, points_d, points_size, n_points_d, sum_x_d, sum_y_d, sum_z_d);
    checkCuda(hipDeviceSynchronize());
    
    //copy points from device to host
    checkCuda(hipMemcpy(points_h, points_d, sizeof(Point)*points_size, hipMemcpyDeviceToHost));  
    checkCuda(hipDeviceSynchronize());
    
    //Free Allocated pointers
    checkCuda(hipFree(points_d));
    checkCuda(hipFree(n_points_d));
    checkCuda(hipFree(sum_x_d));
    checkCuda(hipFree(sum_y_d));
    checkCuda(hipFree(sum_z_d));
    checkCuda(hipDeviceSynchronize());
}

*/
