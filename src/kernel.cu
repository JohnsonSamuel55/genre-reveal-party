#include "hip/hip_runtime.h"
#include <cstdlib>
#include <stdio.h>
#include <float.h>

#include "point.h"

//Testing for CUDA errors
inline hipError_t checkCuda(hipError_t result) {
if (result != hipSuccess) {
fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
}
return result;
}

//Clustering Calculation
__global__ 
void my_Kernel_1(Point* points_d, int points_size, Point* c_d, int cluster_id) {
    
    //Get the index for the current point to work with
    int Col = threadIdx.x + blockIdx.x * blockDim.x;
    int Row = threadIdx.y + blockIdx.y * blockDim.y;
    int index = Col + (Row * blockDim.x *gridDim.x); 
    
    if(index < points_size) {
        Point p = points_d[index];
        double dist = ((c_d->x - p.x) * (c_d->x - p.x) + (c_d->y - p.y) * (c_d->y - p.y) + (c_d->z - p.z) * (c_d->z - p.z));// point_distance(c, *p);
        if (dist < p.min_dist) {
            p.min_dist = dist;
            p.cluster = cluster_id;
        }
        
        points_d[index] = p;
    }
}

__global__
void my_Kernel_2(double max, Point* points_d, int points_size, int* n_points_d, double* sum_x_d, double* sum_y_d, double* sum_z_d) {
    //Get the index of the point to work with
    int Col = threadIdx.x + blockIdx.x * blockDim.x;
    int Row = threadIdx.y + blockIdx.y * blockDim.y;
    int index = Col + (Row * blockDim.x * gridDim.x); 
    
    if(index < points_size) {
        Point p = points_d[index];
        n_points_d[p.cluster] += 1;
        sum_x_d[p.cluster] += p.x;
        sum_y_d[p.cluster] += p.y;
        sum_z_d[p.cluster] += p.z;

        // reset distance
        p.min_dist = max;
        points_d[index] = p;
    }
}

// Function that launches the CUDA kernel
extern "C" void cuda_Kernel_1(Point* points_h, int points_size, Point c, int cluster_id) {
    Point* points_d;
    Point* c_d;
    int new_points_size = sqrt(points_size); 


    //Allocate device pointers and copy them to the device
    checkCuda(hipMalloc((void **) &points_d, sizeof(Point)*points_size));
    checkCuda(hipMalloc((void **) &c_d, sizeof(Point)));
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipMemcpy(points_d, points_h, sizeof(Point)*points_size, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(c_d, &c, sizeof(Point), hipMemcpyHostToDevice));  
    checkCuda(hipDeviceSynchronize());
    
    dim3 DimGrid(ceil(new_points_size/32.0), ceil(new_points_size/32.0), 1);
    dim3 DimBlock(32, 32, 1);

    //Launch kernel
    my_Kernel_1<<<DimGrid, DimBlock>>>(points_d, points_size, c_d, cluster_id);
    checkCuda(hipDeviceSynchronize());

    //copy device points to host points
    checkCuda(hipMemcpy(points_h, points_d, sizeof(Point)*points_size, hipMemcpyDeviceToHost));
    checkCuda(hipDeviceSynchronize());
    
    //Free device pointers
    checkCuda(hipFree(points_d));
    checkCuda(hipFree(c_d));
    checkCuda(hipDeviceSynchronize());
}

// Function that launches the CUDA kernel
extern "C" void cuda_Kernel_2(Point* points_h, int points_size, int* n_points, double* sum_x, double* sum_y, double* sum_z, int k) {
    Point* points_d;
    int* n_points_d;
    double* sum_x_d;
    double* sum_y_d;
    double* sum_z_d;
    int new_points_size = sqrt(points_size); 

    //Allocate device pointers and copy them to the device
    checkCuda(hipMalloc((void **) &points_d, sizeof(Point)*points_size));
    checkCuda(hipMalloc((void **) &n_points_d, sizeof(int)*k));
    checkCuda(hipMalloc((void **) &sum_x_d, sizeof(double)*k));
    checkCuda(hipMalloc((void **) &sum_y_d, sizeof(double)*k));
    checkCuda(hipMalloc((void **) &sum_z_d, sizeof(double)*k));
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipMemcpy(points_d, points_h, sizeof(Point)*points_size, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(n_points_d, n_points, sizeof(int)*k, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(sum_x_d, sum_x, sizeof(double)*k, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(sum_y_d, sum_y, sizeof(double)*k, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(sum_z_d, sum_z, sizeof(double)*k, hipMemcpyHostToDevice));
    checkCuda(hipDeviceSynchronize());
    
    dim3 DimGrid(ceil(new_points_size/32.0), ceil(new_points_size/32.0), 1);
    dim3 DimBlock(32, 32, 1);

    //Launch the kernel
    my_Kernel_2<<<DimGrid, DimBlock>>>(DBL_MAX, points_d, points_size, n_points_d, sum_x_d, sum_y_d, sum_z_d);
    checkCuda(hipDeviceSynchronize());
    
    //copy points from device to host
    checkCuda(hipMemcpy(points_h, points_d, sizeof(Point)*points_size, hipMemcpyDeviceToHost));  
    checkCuda(hipDeviceSynchronize());
    
    //Free Allocated pointers
    checkCuda(hipFree(points_d));
    checkCuda(hipFree(n_points_d));
    checkCuda(hipFree(sum_x_d));
    checkCuda(hipFree(sum_y_d));
    checkCuda(hipFree(sum_z_d));
    checkCuda(hipDeviceSynchronize());
}